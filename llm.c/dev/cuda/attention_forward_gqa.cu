#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#define ENABLE_BF16
#include "common.h"

// ----------------------------------------------------------------------------
// CUDA & cuDNN setup
static bool first_run_validation = true; // always run e.g. permute on 1st run

#ifdef ENABLE_CUDNN
#include <cudnn_frontend.h>
namespace fe = cudnn_frontend;
#if CUBLAS_LOWP == HIP_R_16BF
#define CUDNN_16BIT fe::DataType_t::BFLOAT16
#else
#define CUDNN_16BIT fe::DataType_t::HALF
#endif

static hipdnnHandle_t cudnn_handle;
static size_t cudnn_workspace_size = 0; // dynamically allocated as needed (up to 256MiB!)
static void *cudnn_workspace = NULL;

#define checkCudaErr(err) assert((int)err == 0);
#define checkCudnnErr(err) assert((int)err == 0);
#endif // ENABLE_CUDNN
// ----------------------------------------------------------------------------

void attention_forward_gqa(float *out, float *qkvr, float *att, float *inp,
                           float *freq_cos, float *freq_sin,
                           int B, int T, int C, int NH, int num_kv_heads)
{
    // Note: `inp` is not needed for backward pass, so we re-use it as a scratch buffer.
    // Its contents will be overwritten by this function.
    const int block_size = 256;
    const int softmax_block_size = 256;

    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size
    int queries_per_kv = NH / num_kv_heads;

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    // size_t ksize = sizeof(k) / sizeof(k[0]);
    // size_t vsize = sizeof(v) / sizeof(v[0]);
    // printf("ksize-Vsize: %ld, %ld\n%d, %d, %d\n", ksize, vsize, HS, kv_HS, queries_per_kv);
    int total_threads = B * NH * T * HS;
    int num_blocks = CEIL_DIV(total_threads, block_size);

    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);
    cudaCheck(hipGetLastError());

    apply_rope_forward(q, k, freq_cos, freq_sin, B, T, NH, (C / NH));

    // Repeat interleave for GQA
    if (num_kv_heads != NH)
    {
        float *new_k, *new_v;
        hipMalloc((void **)&new_k, B * NH * T * HS * sizeof(float));
        hipMalloc((void **)&new_v, B * NH * T * HS * sizeof(float));

        int repeat_interleave_threads = B * num_kv_heads * queries_per_kv * T * HS;
        repeat_interleave_forward_kernel<<<num_blocks, block_size>>>(new_k, k, B, num_kv_heads, T, HS, queries_per_kv);
        repeat_interleave_forward_kernel<<<num_blocks, block_size>>>(new_v, v, B, num_kv_heads, T, HS, queries_per_kv);
        cudaCheck(hipGetLastError());

        // Copy the contents of new_k and new_v back to k and v
        hipMemcpy(k, new_k, B * NH * T * HS * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(v, new_v, B * NH * T * HS * sizeof(float), hipMemcpyDeviceToDevice);

        hipFree(new_k);
        hipFree(new_v);
    }

    // size_t k1size = sizeof(k) / sizeof(k[0]);
    // size_t v1size = sizeof(v) / sizeof(v[0]);
    // printf("%ld, %ld", k1size, v1size);

    // Batched matrix multiply with cuBLAS for QK^T
    const float alpha = 1.0f;
    const float beta = 0.0f;
    float *preatt = inp;
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, &alpha, k, HS, T * HS, q, HS, T * HS, &beta, preatt, T, T * T, B * NH));
    // size_t sizepatt = sizeof(preatt) / sizeof(preatt[0]);
    // printf("Preatt: %ld", sizepatt);

    // Multiply all elements of preatt elementwise by scale
    float scale = 1.0 / sqrtf(HS);
    int grid_size = CEIL_DIV(B * NH * T * 32, softmax_block_size);
    softmax_forward_kernel5<<<grid_size, softmax_block_size>>>(att, scale, preatt, B * NH, T);

    cudaCheck(hipGetLastError());

    // New approach: first cuBLAS another batched matmul
    float *vaccum = inp;
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HS, T, T, &alpha, v, HS, T * HS, att, T, T * T, &beta, vaccum, HS, T * HS, B * NH));

    // Now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}