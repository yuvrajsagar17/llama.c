#include "hip/hip_runtime.h"
/*
Kernels for swiglu forward pass.
NOTE: The results shown are performed on L4-GPU

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt swiglu_forward.cu -o swiglu_forward

- version 1 is naive port from CPU code to kernel: parallelizes over B,T,C
./swiglu_forward 1

RESULTS:
block_size   32 | time 0.2018 ms | bandwidth 187.05 GB/s
block_size   64 | time 0.1769 ms | bandwidth 213.37 GB/s
block_size  128 | time 0.1762 ms | bandwidth 214.22 GB/s
block_size  256 | time 0.1762 ms | bandwidth 214.21 GB/s
block_size  512 | time 0.1750 ms | bandwidth 215.73 GB/s
block_size 1024 | time 0.1939 ms | bandwidth 194.65 GB/s

- version 2 uses co-operative groups to work with warp-level reductions with a warp_size of 32 threads, parallelizes over B,T,C
./swiglu_forward 2

RESULTS:
block_size   32 | time 0.1711 ms | bandwidth 220.62 GB/s
block_size   64 | time 0.1694 ms | bandwidth 222.87 GB/s
block_size  128 | time 0.1686 ms | bandwidth 223.90 GB/s
block_size  256 | time 0.1697 ms | bandwidth 222.44 GB/s
block_size  512 | time 0.1689 ms | bandwidth 223.46 GB/s
block_size 1024 | time 0.1701 ms | bandwidth 221.87 GB/s

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ENABLE_BF16
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

void swiglu_forward_cpu(float *out, const float *inp, const float *gate, int N)
{
    /**
     *
     * SwiGLU(x) = Swish(x) * Gate(x)
     * SwiGLU(x) = SiLU(x*W) * (x*V)
     * SiLU is the Swish activation function.
     * inp = x*W
     * gate = x*V
     *
     */

    for (int i = 0; i < N; i++)
    {
        float xW = inp[i];
        float xV = gate[i];
        out[i] = (xW / (1.0f + expf(-xW))) * xV;
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void swiglu_forward_kernel1(floatX *out, const floatX *inp, const floatX *gate, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        floatX xiW = inp[i];
        floatX xiV = gate[i];
        out[i] = (xiW / (floatX)(1.0f + expf(-xiW))) * xiV;
    }
}

/**
 *  `x128` is a custom-datatype that represents a vector of data, typically used to pack multiple scalar values (like floats) into a single register for more efficient parallel processing.
 *  It holds 128 bits of data, which is commonly 4 float values (1 float -> 32 bits => 4*32 = 128)
 *
 *  By loading, processing, and storing 4 floats at a time (or however many the data type holds), you reduce the number of memory accesses and arithmetic operations, and thus resulting in better throughput and efficiency
 */
__global__ void swiglu_forward_kernel2(floatX *out, const floatX *inp, const floatX *gate, int N)
{
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;
    if (i < N)
    {
        x128 packed_out;
        x128 packed_inp = load128cs(inp + i);   // load input (W) without cache streaming
        x128 packed_gate = load128cs(gate + i); // load gate (V) without cache streaming

        for (int k = 0; k < packed_inp.size; ++k)
        {
            floatX xiW = (floatX)packed_inp[k];                        // Extract element from packed input
            floatX xiV = (floatX)packed_gate[k];                       // Extract element from packed gate
            packed_out[k] = (xiW / (floatX)(1.0f + expf(-xiW))) * xiV; // SwiGLU operation
        }

        // Store the result back in memory (cached)
        store128(out + i, packed_out);
    }
}
// ----------------------------------------------------------------------------
// kernel launcher

void swiglu_forward1(floatX *out, const floatX *inp, const floatX *gate, int N, const int block_size)
{
    const int grid_size = ceil_div(N, block_size);
    swiglu_forward_kernel1<<<grid_size, block_size>>>(out, inp, gate, N);
    cudaCheck(hipGetLastError());
}

void swiglu_forward2(floatX *out, const floatX *inp, const floatX *gate, int N, const int block_size)
{
    const int grid_size = ceil_div(N, block_size * x128::size);
    swiglu_forward_kernel2<<<grid_size, block_size>>>(out, inp, gate, N);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void swiglu_forward(int kernel_num,
                    floatX *out, const floatX *inp, const floatX *gate,
                    int B, int T, int C, int block_size)
{
    switch (kernel_num)
    {
    case 1:
        swiglu_forward1(out, inp, gate, B * T * C, block_size);
        break;
    case 2:
        swiglu_forward2(out, inp, gate, B * T * C, block_size);
        break;

    default:
        printf("Invalid kernel number\n");
        exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, const char **argv)
{
    setup_main();

    int B = 8;
    int T = 1024;
    int C = 768;

    // create host memory of random numbers
    float *out = (float *)malloc(B * T * C * sizeof(float));
    float *inp = make_random_float(B * T * C);  // precomputed x*W
    float *gate = make_random_float(B * T * C); // precomputed x*V

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1)
    {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    swiglu_forward_cpu(out, inp, gate, B * T * C);

    // move to GPU
    floatX *d_out;
    floatX *d_inp;
    floatX *d_gate; // Allocate device memory for gate
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_gate, B * T * C * sizeof(floatX))); // Allocate device memory for gate
    cudaCheck(memcpy_convert(d_inp, inp, B * T * C));
    cudaCheck(memcpy_convert(d_gate, gate, B * T * C)); // Copy gate data to device

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++)
    {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        swiglu_forward(kernel_num, d_out, d_inp, d_gate, B, T, C, block_size); // Pass gate to the kernel
#if !defined(ENABLE_BF16) && !defined(ENABLE_FP16)
        float tol = 1e-5;
#else
        float tol = 1e-2f;
#endif
        validate_result(d_out, out, "out", B * T * C, tol);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++)
    {
        int block_size = block_sizes[j];

        int repeat_times = 1000;

        float elapsed_time = benchmark_kernel(repeat_times, swiglu_forward,
                                              kernel_num, d_out, d_inp, d_gate,
                                              B, T, C, block_size); // Pass gate to the benchmark

        // napkin math: estimate the memory bandwidth achieved
        // for each (B,T,C) output element, we do 1 read and 1 write, 4 bytes each
        // and e.g. A100 40GB PCIe is advertised at 1,555GB/s
        long memory_ops = B * T * C * 3 * (int)sizeof(floatX); // Include gate in memory operations
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    free(out);
    free(inp);
    free(gate); // Free host memory for gate

    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_gate)); // Free device memory for gate
    return 0;
}
