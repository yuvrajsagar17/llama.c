#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#define ENABLE_BF16
#include "common.h"

// ----------------------------------------------------------------------------
// CUDA & cuDNN setup
static bool first_run_validation = true; // always run e.g. permute on 1st run

#ifdef ENABLE_CUDNN
#include <cudnn_frontend.h>
namespace fe = cudnn_frontend;
#if CUBLAS_LOWP == HIP_R_16BF
#define CUDNN_16BIT fe::DataType_t::BFLOAT16
#else
#define CUDNN_16BIT fe::DataType_t::HALF
#endif

static hipdnnHandle_t cudnn_handle;
static size_t cudnn_workspace_size = 0; // dynamically allocated as needed (up to 256MiB!)
static void *cudnn_workspace = NULL;

#define checkCudaErr(err) assert((int)err == 0);
#define checkCudnnErr(err) assert((int)err == 0);
#endif // ENABLE_CUDNN
// ----------------------------------------------------------------------------

void attention_backward_gqa(float *dinp, float *dqkvr, float *dpreatt, float *datt,
                            float *scratch,
                            const float *dout,
                            const float *freq_cos, const float *freq_sin,
                            const float *qkvr, const float *att,
                            int B, int T, int C, int NH, int num_kv_heads)
{
    const int block_size = 256;
    int HS = C / NH; // head size
    int queries_per_kv = NH / num_kv_heads;
    const float one = 1.0f;
    const float zero = 0.0f; // note beta = 1.0f so that we accumulate gradients (+=)
    // unpack convenience pointers into q, k, v
    const float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    float *dq, *dk, *dv;
    dq = dqkvr + 0 * B * T * C;
    dk = dqkvr + 1 * B * T * C;
    dv = dqkvr + 2 * B * T * C;

    // backward through the unpermute operation
    int num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel_backward<<<num_blocks, block_size>>>(scratch, dout, B, T, NH, HS);
    cudaCheck(hipGetLastError());

    // backward into datt
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, &one, v, HS, T * HS, scratch, HS, T * HS, &zero, datt, T, T * T, B * NH));

    // backward into dv
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, &one, scratch, HS, T * HS, att, T, T * T, &zero, dv, HS, T * HS, B * NH));

    // backward into preatt
    int hs = C / NH; // head size
    float scale = 1.0f / sqrtf(hs);
    softmax_autoregressive_backward_kernel<<<dim3(T / 4, B * NH), 256>>>(dpreatt, datt, att, B, T, C, scale);
    cudaCheck(hipGetLastError());

    // backward into q
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HS, T, T, &one, k, HS, T * HS, dpreatt, T, T * T, &zero, dq, HS, T * HS, B * NH));

    // backward into k
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, &one, q, HS, T * HS, dpreatt, T, T * T, &zero, dk, HS, T * HS, B * NH));

    // Repeat interleave for GQA if num_kv_heads != NH
    if (num_kv_heads != NH)
    {
        // Allocate intermediate tensors for backward repeat interleave
        float *dsrc_k, *dsrc_v;
        hipMalloc((void **)&dsrc_k, B * num_kv_heads * queries_per_kv * T * HS * sizeof(float));
        hipMalloc((void **)&dsrc_v, B * num_kv_heads * queries_per_kv * T * HS * sizeof(float));
        hipMemset(dsrc_k, 0, B * num_kv_heads * queries_per_kv * T * HS * sizeof(float));
        hipMemset(dsrc_v, 0, B * num_kv_heads * queries_per_kv * T * HS * sizeof(float));

        // backward through repeat interleave operation for dk and dv
        int repeat_interleave_threads = B * NH * T * HS;
        num_blocks = CEIL_DIV(repeat_interleave_threads, block_size);
        repeat_interleave_backward_kernel<<<num_blocks, block_size>>>(dsrc_k, dk, B, num_kv_heads, T, HS, queries_per_kv);
        repeat_interleave_backward_kernel<<<num_blocks, block_size>>>(dsrc_v, dv, B, num_kv_heads, T, HS, queries_per_kv);
        cudaCheck(hipGetLastError());

        // Apply RoPE backward
        apply_rope_backward(dq, dsrc_k, q, k, freq_cos, freq_sin, B, T, NH, (C / NH)); // (C /NH) = (C /NH) is the head_dim (hs)

        // backward into inp
        num_blocks = CEIL_DIV(B * NH * T * HS, block_size);
        permute_kernel_backward<<<num_blocks, block_size>>>(dinp, dq, dsrc_k, dsrc_v, B, T, NH, HS);
        cudaCheck(hipGetLastError());

        // Cleanup
        hipFree(dsrc_k);
        hipFree(dsrc_v);
    }
    else
    {
        // backward into inp
        // backward into inp without repeat interleave
        num_blocks = CEIL_DIV(B * NH * T * HS, block_size);
        permute_kernel_backward<<<num_blocks, block_size>>>(dinp, dq, dk, dv, B, T, NH, HS);
        cudaCheck(hipGetLastError());
    }
}