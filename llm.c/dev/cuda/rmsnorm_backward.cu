#include "hip/hip_runtime.h"
/*
Kernels for layernorm backward pass.

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt layernorm_backward.cu -o layernorm_backward

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops over C
./layernorm_backward 1

version 2 moves a lot of reduction to shared memory over global memory
./layernorm_backward 2
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#define ENABLE_BF16
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

// LLama RMSNorm forward pass
void rmsnorm_forward_cpu(float *out, const float *inp, const float *weight, const float *bias, int B, int T, int C)
{
    float eps = 1e-5f;
    for (int b = 0; b < B; b++)
    {
        for (int t = 0; t < T; t++)
        {
            // seek to the input position inp[b,t,:]
            const float *x = inp + b * T * C + t * C;
            // calculate the rms (root mean square)
            float rms = 0.0f;
            for (int i = 0; i < C; i++)
            {
                rms += x[i] * x[i];
            }
            rms = sqrtf(rms / C + eps);
            // seek to the output position in out[b,t,:]
            float *out_bt = out + b * T * C + t * C;
            for (int i = 0; i < C; i++)
            {
                float n = x[i] / rms;              // normalized output
                float o = n * weight[i] + bias[i]; // scale and shift it
                out_bt[i] = o;                     // write
            }
        }
    }
}

void rmsnorm_backward_cpu(float *dinp, float *dweight, float *dbias,
                          const float *dout, const float *inp, const float *weight, const float *bias,
                          int B, int T, int C)
{
    float eps = 1e-5f;
    for (int b = 0; b < B; b++)
    {
        for (int t = 0; t < T; t++)
        {
            const float *dout_bt = dout + b * T * C + t * C;
            const float *inp_bt = inp + b * T * C + t * C;
            float *dinp_bt = dinp + b * T * C + t * C;

            // Calculate the rms
            float rms = 0.0f;
            for (int i = 0; i < C; i++)
            {
                rms += inp_bt[i] * inp_bt[i];
            }
            rms = sqrtf(rms / C + eps);

            // First, calculate the gradients for the weights and biases
            for (int i = 0; i < C; i++)
            {
                float norm = inp_bt[i] / rms;
                dbias[i] += dout_bt[i];
                dweight[i] += norm * dout_bt[i];
            }

            // Now, calculate the gradients for the inputs
            float drms = 0.0f;
            for (int i = 0; i < C; i++)
            {
                drms += inp_bt[i] * dout_bt[i] * weight[i];
            }
            drms = drms * (-1.0f / (rms * rms * rms * C));

            for (int i = 0; i < C; i++)
            {
                float norm = inp_bt[i] / rms;
                dinp_bt[i] = dout_bt[i] * weight[i] / rms + drms * inp_bt[i];
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// GPU helper functions for atomicAdd on smaller than 32-bit types
#ifdef ENABLE_BF16
__device__ void atomicAddX(__hip_bfloat16 *addr, __hip_bfloat16 val)
{
    uintptr_t ptr_val = reinterpret_cast<uintptr_t>(addr);
    __hip_bfloat162 *ptr_bf16 = reinterpret_cast<__hip_bfloat162 *>(ptr_val & ~uintptr_t(0x3));

    // Prepare the value to add, setting the other half to zero
    __hip_bfloat162 add_val = (ptr_val & 0x3) ? __halves2bfloat162(__ushort_as_bfloat16(0), val)
                                             : __halves2bfloat162(val, __ushort_as_bfloat16(0));
    atomicAdd(ptr_bf16, add_val);
}
#endif
#ifdef ENABLE_FP16
__device__ void atomicAddX(half *addr, half val)
{
    uintptr_t ptr_val = reinterpret_cast<uintptr_t>(addr);
    half2 *ptr_fp16 = reinterpret_cast<half2 *>(ptr_val & ~uintptr_t(0x3));

    // Prepare the value to add, setting the other half to zero
    half2 add_val = (ptr_val & 0x3) ? __halves2half2(__ushort_as_half(0), val)
                                    : __halves2half2(val, __ushort_as_half(0));
    atomicAdd(ptr_fp16, add_val);
}
#endif
__device__ void atomicAddX(float *addr, float val)
{
    atomicAdd(addr, val);
}

__global__ void rmsnorm_backward_kernel1(float *dinp, float *dweight, float *dbias,
                                         const float *dout, const float *inp, const float *weight, const float *bias,
                                         int N, int C)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    float eps = 1e-5f;
    const float *dout_bt = dout + idx * C;
    const float *inp_bt = inp + idx * C;
    float *dinp_bt = dinp + idx * C;

    // Calculate the rms
    float rms = 0.0f;
    for (int i = 0; i < C; i++)
    {
        rms += inp_bt[i] * inp_bt[i];
    }
    rms = sqrtf(rms / C + eps);

    // First, calculate the gradients for the weights and biases
    for (int i = 0; i < C; i++)
    {
        float norm = inp_bt[i] / rms;
        atomicAdd(&dbias[i], dout_bt[i]);
        atomicAdd(&dweight[i], norm * dout_bt[i]);
    }

    // Calculate drms
    float drms = 0.0f;
    for (int i = 0; i < C; i++)
    {
        drms += inp_bt[i] * dout_bt[i] * weight[i];
    }
    drms = drms * (-1.0f / (rms * rms * rms * C));

    // Now, calculate the gradients for the inputs
    for (int i = 0; i < C; i++)
    {
        float norm = inp_bt[i] / rms;
        dinp_bt[i] = dout_bt[i] * weight[i] / rms + drms * inp_bt[i];
    }
}

// ----------------------------------------------------------------------------
// kernel launchers

void rmsnorm_backward1(float *dinp, float *dweight, float *dbias,
                       const float *dout, const float *inp, const float *weight, const float *bias,
                       int B, int T, int C, const int block_size)
{
    const int N = B * T;
    const int grid_size = (N + block_size - 1) / block_size; // equivalent to ceil(N / block_size)
    rmsnorm_backward_kernel1<<<grid_size, block_size>>>(dinp, dweight, dbias, dout, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void rmsnorm_backward(int kernel_num,
                      floatX *dinp, floatX *dweight, floatX *dbias,
                      const floatX *dout, const floatX *inp, const floatX *weight, const floatX *bias,
                      int B, int T, int C,
                      const int block_size)
{
    switch (kernel_num)
    {
    case 1:
        rmsnorm_backward1(dinp, dweight, dbias, dout, inp, weight, bias, B, T, C, block_size);
        break;

    default:
        printf("Invalid kernel number\n");
        exit(1);
    }
    cudaCheck(hipGetLastError());
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv)
{
    setup_main();

    int B = 8;
    int T = 1024;
    int C = 1600; // this is the problematic size

    // first do the forward pass in CPU
    float *out = (float *)malloc(B * T * C * sizeof(float));

    float *inp = make_random_float(B * T * C);
    float *weight = make_random_float(C);
    float *bias = make_random_float(C);

    rmsnorm_forward_cpu(out, inp, weight, bias, B, T, C);

    // now do the backward pass, again on CPU
    float *dout = make_random_float(B * T * C);
    float *dinp = make_zeros_float(B * T * C);
    float *dweight = make_zeros_float(C);
    float *dbias = make_zeros_float(C);

    rmsnorm_backward_cpu(dinp, dweight, dbias, dout, inp, weight, bias, B, T, C);

    // the above calculations act as the reference
    // now let's do the same on the GPU

    // read kernel_num from command line
    int kernel_num = 2;
    if (argc > 1)
    {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // move all the variables we need for backward pass onto the GPU
    floatX *d_dinp;
    floatX *d_dweight;
    floatX *d_dbias;
    floatX *d_dout;
    floatX *d_inp;
    floatX *d_weight;
    floatX *d_bias;

    float *d_scratch;
    cudaCheck(hipMalloc(&d_dinp, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_dweight, C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_dbias, C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_dout, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_weight, C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_scratch, (1024 / 32) * cuda_num_SMs * (2 * C + 1) * sizeof(float)));
    // copy over the "inputs" to the backward call
    cudaCheck(memcpy_convert(d_dout, dout, B * T * C));
    cudaCheck(memcpy_convert(d_inp, inp, B * T * C));
    cudaCheck(memcpy_convert(d_weight, weight, C));
    cudaCheck(memcpy_convert(d_bias, bias, C));

    // launch the kernel
    // removed 768 because it doesn't work for kernel9 despite being OK in train_gpt2.cu?!
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++)
    {
        int block_size = block_sizes[j];
        // init the "outputs" of the backward call to zeros
        cudaCheck(hipMemset(d_dinp, 0, B * T * C * sizeof(floatX)));
        cudaCheck(hipMemset(d_dweight, 0, C * sizeof(floatX)));
        cudaCheck(hipMemset(d_dbias, 0, C * sizeof(floatX)));

        rmsnorm_backward(kernel_num, d_dinp, d_dweight, d_dbias, d_dout, d_inp, d_weight, d_bias,
                         B, T, C, block_size);

        // check the correctness of the kernel
        float error_threshold_dinp = sizeof(floatX) == 4 ? 1e-3f : 1e-1f;    // allow larger errors for BF16/FP16
        float error_threshold_dparams = sizeof(floatX) == 4 ? 1e-3f : 5e-1f; // much, much larger...
        printf("Checking correctness...\n");
        printf("dinp:\n");
        validate_result(d_dinp, dinp, "dinp", B * T * C, error_threshold_dinp);
        printf("dweight:\n");
        validate_result(d_dweight, dweight, "dweight", C, error_threshold_dparams);
        printf("dbias:\n");
        validate_result(d_dbias, dbias, "dbias", C, error_threshold_dparams);

        printf("All results match for block_size=%d.\n\n", block_size);
    }

    // now time the kernel
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++)
    {
        int block_size = block_sizes[j];
        int repeat_times = 100;
        float elapsed_time = benchmark_kernel(repeat_times, layernorm_backward, kernel_num,
                                              d_dinp, d_dweight, d_dbias, d_scratch, d_dout, d_inp, d_weight, d_mean, d_rstd,
                                              B, T, C, block_size);
        printf("block_size %4d time %.4f ms\n", block_size, elapsed_time);
    }

    // cleanups
    free(out);

    free(inp);
    free(weight);
    free(bias);
    free(dout);
    free(dinp);
    free(dweight);
    free(dbias);
    cudaCheck(hipFree(d_dinp));
    cudaCheck(hipFree(d_dweight));
    cudaCheck(hipFree(d_dbias));
    cudaCheck(hipFree(d_dout));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));
    cudaCheck(hipFree(d_scratch));
    return 0;
}
